#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *       Filename:  cuda_knn_thrust.cpp
 *    Description:  
 *        Created:  2015-02-09 10:30
 *         Author:  Tiago Lobato Gimenes        (tlgimenes@gmail.com)
 * =====================================================================================
 */

////////////////////////////////////////////////////////////////////////////////////////

#include "cuda_knn_thrust.hpp"

#include <hip/hip_runtime.h>
#include <thrust/sort.h>

////////////////////////////////////////////////////////////////////////////////////////

/*
 * @arg0: raw data in the device
 * @arg1: space dimentions
 * @arg2: DIRECT index of the query in the raw data
 * @arg3: return array containing distances from each data to query 
 * */
template <typename T>
__global__ void __comp_dist(T* dev_data, uint dim, uint query, struct sort_t<T> dev_sort)
{
    uint k = blockDim.x * blockIdx.x + threadIdx.x;
    T res_local = 0;
    T res_query_local = 0;
    T res_index_local = 0;

    uint index = k * dim;

    dev_sort._value[k] = k;
    
    for(uint i=0; i < dim; i++)
    {
        res_query_local = dev_data[query + i];
        res_index_local = dev_data[index + i];
        res_local += (res_query_local - res_index_local) * (res_query_local - res_index_local);
    }

    dev_sort._key[k] = res_local;
}

////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ void CUDAKNNThrust<T>::find(uint query, uint k, std::vector<uint>& knn)
{
    int minGridSize, blockPerGrid;
    int threadsPerBlock;
    int N_threads = this->_data.size() / this->_dim;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &threadsPerBlock, 
            (void*)__comp_dist<T>, 0, N_threads);

    blockPerGrid = (N_threads + threadsPerBlock - 1) / threadsPerBlock;

    query = query * this->_dim;

    __comp_dist<T><<<blockPerGrid, threadsPerBlock>>>(this->_dev_data, this->_dim, query, 
            this->_dev_sort);
    CUDA_ERR(hipGetLastError());

    thrust::device_ptr<T> key(this->_dev_sort._key);
    thrust::device_ptr<uint> value(this->_dev_sort._value);
    thrust::sort_by_key(key, key + N_threads, value);
    
    knn.resize(k);
    CUDA_ERR(hipMemcpy(knn.data(), this->_dev_sort._value + 1, k*sizeof(uint), 
                hipMemcpyDeviceToHost));

    return;
}

////////////////////////////////////////////////////////////////////////////////////////

